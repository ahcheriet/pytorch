
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialUpSamplingBicubic.cu"
#else

#include <THCUNN/upsampling.h>
#include <ATen/cuda/CUDAContext.h>

static inline void THNN_(SpatialUpSamplingBicubic_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputHeight, int inputWidth,
                         int outputHeight, int outputWidth) {
  THArgCheck(inputHeight > 0 && inputWidth > 0
             && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (H: %d, W: %d) output (H: %d, W: %d)",
             inputHeight, inputWidth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, !input->is_empty() && input->dim() == 4, 2, input,
                     "non-empty 4D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, outputWidth);
  }
}

void THNN_(SpatialUpSamplingBicubic_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputHeight = THCTensor_(size)(state, input, 2);
  int inputWidth = THCTensor_(size)(state, input, 3);
  THNN_(SpatialUpSamplingBicubic_shapeCheck)
       (state, input, NULL,
        nbatch, channels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);

  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resize4d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputHeight, outputWidth);
  THCTensor_(zero)(state, output);
  THCDeviceTensor<scalar_t, 4> idata = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> odata = toDeviceTensor<scalar_t, 4>(state, output);
  THAssert(inputHeight > 0 && inputWidth > 0 && outputHeight > 0 && outputWidth > 0);

  // Get scaling factors
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);

  const int num_output_elements = outputHeight * outputWidth;
  const int max_threads =
    at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;

  // Launch kernel
  cudaStream_t stream = THCState_getCurrentStream(state);
  bicubic_interp2d_kernel<scalar_t, accreal> <<<
    THCCeilDiv(num_output_elements, max_threads),
    max_threads,
    0,
    stream
  >>>(num_output_elements, rheight, rwidth, idata, odata);
  THCudaCheck(cudaGetLastError());
}


void THNN_(SpatialUpSamplingBicubic_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputHeight,
           int inputWidth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  THNN_(SpatialUpSamplingBicubic_shapeCheck)
       (state, NULL, gradOutput,
        nbatch, nchannels,
        inputHeight, inputWidth,
        outputHeight, outputWidth);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THCTensor_(resize4d)(state, gradInput, nbatch, nchannels, inputHeight, inputWidth);
  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<scalar_t, 4> in_data = toDeviceTensor<scalar_t, 4>(state, gradInput);
  THCDeviceTensor<scalar_t, 4> out_data = toDeviceTensor<scalar_t, 4>(state, gradOutput);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputHeight * outputWidth;
  const int num_threads =
    at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  cudaStream_t stream = THCState_getCurrentStream(state);
  bicubic_interp2d_backward_kernel<scalar_t ,accreal> <<<THCCeilDiv(num_kernels, num_threads),
  num_threads, 0, stream>>>(num_kernels, rheight, rwidth, align_corners, in_data, out_data);
  THCudaCheck(cudaGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif
